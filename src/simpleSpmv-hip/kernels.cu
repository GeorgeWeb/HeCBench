#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <chrono>
#include <hip/hip_runtime.h>
#include "mv.h"

// sparse matrix vector multiply using the CSR format
__global__ void mv_csr(const size_t num_rows,
                       const size_t *row_indices,
                       const size_t *col_indices,
                       const REAL *values,
                       const REAL *x,
                             REAL *y)
{
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < num_rows) {
    size_t row_start = row_indices[i];
    size_t row_end = row_indices[i+1];

    REAL temp = 0;
    for(size_t n = row_start; n < row_end; n++){
      temp += values[n] * x[col_indices[n]];
    }
    y[i] = temp;
  }
}

// dense matrix vector multiply
__global__ void mv_dense(const size_t num_rows, const REAL* matrix, const REAL* x, REAL* y)
{
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < num_rows) {
    REAL temp = 0;
    for (size_t j = 0; j < num_rows; j++) {
      if (matrix[i * num_rows + j] != (REAL)0) 
        temp += matrix[i * num_rows + j] * x[j];
    }
    y[i] = temp;
  }
}

long mv_dense_parallel(const int repeat,
                       const int bs,
                       const size_t num_rows,
                       const REAL* x,
                             REAL* matrix,
                             REAL* y)
{
  REAL *d_x, *d_matrix, *d_y;
  hipMalloc(&d_x, num_rows*sizeof(REAL));
  hipMalloc(&d_matrix, num_rows * num_rows * sizeof(REAL));
  hipMalloc(&d_y, num_rows*sizeof(REAL));

  hipMemcpy(d_x, x, num_rows*sizeof(REAL), hipMemcpyHostToDevice);
  hipMemcpy(d_matrix, matrix, num_rows*num_rows*sizeof(REAL), hipMemcpyHostToDevice);

  dim3 grids ((num_rows + bs - 1) / bs);
  dim3 blocks (bs);

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++)
    mv_dense<<<grids, blocks>>>(num_rows, d_matrix, d_x, d_y);

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  hipMemcpy(y, d_y, num_rows*sizeof(REAL), hipMemcpyDeviceToHost);

  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_matrix);

  return time;
}

long mv_csr_parallel(const int repeat,
                     const int bs,
                     const size_t num_rows,
                     const size_t *row_indices,
                     const size_t *col_indices,
                     const REAL* values,
                     const REAL* x,
                     const size_t nnz,
                     REAL* matrix,
                     REAL* y)
{
  size_t *d_row_indices;
  size_t *d_col_indices;
  REAL *d_values, *d_x, *d_y;

  hipMalloc(&d_row_indices, (num_rows+1)*sizeof(size_t));
  hipMalloc(&d_col_indices, nnz*sizeof(size_t));
  hipMalloc(&d_values, nnz*sizeof(REAL));
  hipMalloc(&d_x, num_rows*sizeof(REAL));
  hipMalloc(&d_y, num_rows*sizeof(REAL));

  hipMemcpy(d_row_indices, row_indices, (num_rows+1)*sizeof(size_t), hipMemcpyHostToDevice);
  hipMemcpy(d_col_indices, col_indices, nnz*sizeof(size_t), hipMemcpyHostToDevice);
  hipMemcpy(d_values, values, nnz*sizeof(REAL), hipMemcpyHostToDevice);
  hipMemcpy(d_x, x, num_rows*sizeof(REAL), hipMemcpyHostToDevice);

  dim3 grids ((num_rows + bs - 1) / bs);
  dim3 blocks (bs);

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++)
    mv_csr<<<grids, blocks>>>(num_rows, d_row_indices, d_col_indices, d_values, d_x, d_y);

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();

  hipMemcpy(y, d_y, num_rows*sizeof(REAL), hipMemcpyDeviceToHost);

  hipFree(d_row_indices);
  hipFree(d_col_indices);
  hipFree(d_values);
  hipFree(d_x);
  hipFree(d_y);

  return time;
}
