#include "hip/hip_runtime.h"
#include <chrono>
#include <stdio.h>
#include <hip/hip_runtime.h>

// reference
// https://stackoverflow.com/questions/59879285/whats-the-alternative-for-match-any-sync-on-compute-capability-6

// increment the value at ptr by 1 and return the old value
#define warpSize 32

__device__ int atomicAggInc(int* ptr) {
  int mask;
  for (int i = 0; i < warpSize; i++){
    unsigned long long tptr = __shfl((unsigned long long)ptr, i);
    unsigned my_mask = __ballot((tptr == (unsigned long long)ptr));
    if (i == (threadIdx.x & (warpSize-1))) mask = my_mask;
  }
  int leader = __ffs(mask) - 1;  // select a leader
  int res = 0;
  unsigned lane_id = threadIdx.x % warpSize;
  if (lane_id == leader) {                 // leader does the update
    res = atomicAdd(ptr, __popc(mask));
  }
  res = __shfl(res, leader);    // get leader’s old value
  return res + __popc(mask & ((1 << lane_id) - 1)); //compute old value
}

__global__ void k(int *d) {
  int *ptr = d + threadIdx.x % 32;
  atomicAggInc(ptr);
}

const int ds = 32;

int main(int argc, char* argv[]) {
  if (argc != 2) {
    printf("Usage: %s <repeat>\n", argv[0]);
    return 1;
  }
  const int repeat = atoi(argv[1]);

  int *d_d, *h_d;
  h_d = new int[ds];
  hipMalloc(&d_d, ds*sizeof(d_d[0]));
  hipMemset(d_d, 0, ds*sizeof(d_d[0]));

  hipDeviceSynchronize();
  
  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++)
    hipLaunchKernelGGL(k, 256*32, 256, 0, 0, d_d);
  hipDeviceSynchronize();

  auto end = std::chrono::steady_clock::now();
  std::chrono::duration<float> time = end - start;
  printf("Total kernel time: %f (s)\n", time.count());

  hipMemcpy(h_d, d_d, ds*sizeof(d_d[0]), hipMemcpyDeviceToHost);

  bool ok = true;
  for (int i = 0; i < ds; i++) {
    if (h_d[i] != 256 * 256 * repeat) {
      ok = false;
      break;
    }
  }
  printf("%s\n", ok ? "PASS" : "FAIL");
  hipFree(d_d);
  delete [] h_d;
  return 0;
}
