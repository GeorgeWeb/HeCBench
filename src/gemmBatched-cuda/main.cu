#include <assert.h>
#include <stdlib.h>
#include <unistd.h>
#include <chrono>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>

using namespace std;

int main(int argc, char ** argv){

  int status;
  int lower = 2;    // lower bound to the matrix dimension
  int upper = 100;  // upper bound to the matrix dimension
  int num = 25000;  // batch size
  int reps = 10;
  int verbose = 0;
  
  while((status = getopt(argc, argv, "l:u:n:r:v")) != -1){
    switch(status){
    case 'l':
      lower = strtoul(optarg, 0, 0);
      break;
    case 'u':
      upper = strtoul(optarg, 0, 0);
      break;
    case 'n':
      num = strtoul(optarg, 0, 0);  // batch size
      break;
    case 'r':
      reps = strtoul(optarg, 0, 0);
      break;
    case 'v':
      verbose = 1;
      break;
    default:
      cerr << "invalid argument: " << status << endl;
      exit(1);
    }
  }

  cout << "running with" << " lower: " << lower << " upper: " << upper
       << " num: " << num << " reps: " << reps << endl;

  if(verbose) cout << "initializing inputs" << endl;
  size_t matrices_size = upper * upper * num * sizeof(float);
  size_t vectors_size = upper * num * sizeof(float);

  float *matrices = (float*)malloc(matrices_size);
  assert(matrices);

  float *vectors = (float*)malloc(vectors_size);
  assert(vectors);

  srand48(48);
  for(int i = 0; i < num * upper * upper; i++)
    matrices[i] = drand48();

  for(int i = 0; i < num * upper; i++)
    vectors[i] = drand48();

  hipError_t cudaStat;
  hipblasStatus_t stat;
  hipblasHandle_t handle;

  stat = hipblasCreate(&handle);
  if(stat != HIPBLAS_STATUS_SUCCESS){
    cerr << "cublas init failed" << endl;
    exit(1);
  }

  if(verbose) cout << "allocating device variables" << endl;

  // allocate input space on device
  float *devMatrices;
  cudaStat = hipMalloc((void**)&devMatrices, matrices_size);
  assert(!cudaStat);

  float *devVectors;
  cudaStat = hipMalloc((void**)&devVectors, vectors_size);
  assert(!cudaStat);

  // allocate result space on device
  float *devResult;
  cudaStat = hipMalloc((void**)&devResult, vectors_size);

  assert(!cudaStat);

  if(verbose) cout << "copying data to device" << endl;
  // copy data to device
  cudaStat = 
    hipMemcpy(devMatrices, matrices, matrices_size, hipMemcpyHostToDevice);

  assert(!cudaStat);
  
  cudaStat = 
    hipMemcpy(devVectors, vectors, vectors_size, hipMemcpyHostToDevice);

  assert(!cudaStat);

  // create lists of device pointers to inputs and outputs
  float **AList = 0, **BList = 0, **CList = 0;

  AList = (float**)malloc(num * sizeof(float*));
  BList = (float**)malloc(num * sizeof(float*));
  CList = (float**)malloc(num * sizeof(float*));

  int lda = upper, // lda >= max(1,m)
      ldb = upper, // ldb >= max(1,k)
      ldc = upper; // ldc >= max(1,m)

  const float alpha = 1.0f, beta = 0.0f;
  for(int i = 0; i < num; i++){
    // each array of dim. lda x k
    AList[i] = devMatrices + upper * upper * i;
    // each array of dim. ldb x n
    BList[i] = devVectors + upper * i;
    // each array of dim. ldc x n
    CList[i] = devResult + upper * i;
  }

  // copy pointer lists to device
  float **devAList, **devBList, **devCList;
  cudaStat = hipMalloc((void**)&devAList, num * sizeof(float*));
  assert(!cudaStat);

  cudaStat = hipMalloc((void**)&devBList, num * sizeof(float*));
  assert(!cudaStat);

  cudaStat = hipMalloc((void**)&devCList, num * sizeof(float*));
  assert(!cudaStat);

  cudaStat = hipMemcpy(devAList, AList, num * sizeof(float*), hipMemcpyHostToDevice);
  assert(!cudaStat);
  
  cudaStat = hipMemcpy(devBList, BList, num * sizeof(float*), hipMemcpyHostToDevice);
  assert(!cudaStat);

  cudaStat = hipMemcpy(devCList, CList, num * sizeof(float*), hipMemcpyHostToDevice);
  assert(!cudaStat);


  /* perform <num> <size x size> x <size x 1> multiplications 
     with distinct matrices
   */
  for(int size = lower; size <= upper; size++){
    if(verbose) cout << "running with <size x size> x <size x 1> " << size << endl;
    double sum = 0.0;
    const int m = size, n = 1, k = size;
    for(int rep = 0; rep <= reps; rep++){
      auto start = std::chrono::steady_clock::now();
      stat = hipblasSgemmBatched(handle,
                                HIPBLAS_OP_N,
                                HIPBLAS_OP_N,
                                m, n, k,
                                &alpha,
                                (const float**)devAList,
                                lda,
                                (const float**)devBList,
                                ldb,
                                &beta,
                                devCList,
                                ldc,
                                num);
      hipDeviceSynchronize();
      auto end = std::chrono::steady_clock::now();
      auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
      auto elapsed = time * 1e-3;

      if(stat != HIPBLAS_STATUS_SUCCESS){
	cerr << "hipblasSgemmBatched failed" << endl;
        break;
      }

      if (rep != 0) sum += elapsed;
      
      if(verbose)
	cout << "size " << size << ": " << elapsed << " us; " 
	     << elapsed / num << " us per operation" << endl;
    }
    cout << "size " << size << " average execution time: " << sum/reps << " us; "
	 << sum / reps / num << " us per operation" << endl;
  }

  hipFree(devMatrices);
  hipFree(devVectors);
  hipFree(devResult);
  hipFree(devAList);
  hipFree(devBList);
  hipFree(devCList);

  free(matrices);
  free(vectors);
  free(AList);
  free(BList);
  free(CList);
      
  return 0;
}
